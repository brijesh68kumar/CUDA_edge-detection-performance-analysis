#include "hip/hip_runtime.h"
#include "wb.h"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <sys/time.h>

using namespace cv;
using namespace std;

#define MASK_SIZE 5
#define sigma 0.9
#define MASK_RADIUS MASK_SIZE/ 2
#define TILE_WIDTH 16
#define SIZE        (TILE_WIDTH + MASK_SIZE - 1)
#define PI 3.141592653589793238  

__constant__ float M[MASK_SIZE * MASK_SIZE];

__global__ void convolution2D (float * I,float * P,
        int channels, int width, int height)
{
    __shared__ float N_ds[SIZE][SIZE];

    int bx = blockIdx.x,  by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

      for(int k=0;k<channels;k++){
        int dest  = ty * TILE_WIDTH + tx;
        int destX = dest % SIZE;
        int destY = dest / SIZE;
        int srcY  = by * TILE_WIDTH + destY - MASK_RADIUS;
        int srcX  = bx * TILE_WIDTH + destX - MASK_RADIUS;
        int src   = (srcY * width + srcX) * channels + k;

        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
        else
            N_ds[destY][destX] = 0.0;

        __syncthreads();

        dest  = ty * TILE_WIDTH + tx + TILE_WIDTH * TILE_WIDTH;
        destY = dest / SIZE;
        destX = dest % SIZE;
        srcY  = by * TILE_WIDTH + destY - MASK_RADIUS;
        srcX  = bx * TILE_WIDTH + destX - MASK_RADIUS;
        src   = (srcY * width + srcX) * channels + k;

        if (destY < SIZE) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = I[src];
            else
                N_ds[destY][destX] = 0.0;
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < MASK_SIZE; ++y)
            for (int x = 0; x < MASK_SIZE; ++x)
                accum += N_ds[ty + y][tx + x] * M[y * MASK_SIZE + x];

        int y = by * TILE_WIDTH + ty;
        int x = bx * TILE_WIDTH + tx;
        if (y < height && x < width)
            P[(y * width + x) * channels +k ] = min(max(accum, 0.0), 1.0);

        __syncthreads();
   }
}

int main (int argc, char * argv[ ])
{
    wbArg_t arg;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float hostMaskData[MASK_SIZE*MASK_SIZE];
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;
    clock_t begin = clock();

    clock_t begin_imread = clock();
    arg = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(arg, 0);


    inputImage = wbImport(inputImageFile);
   
    printf("Image Dimension: %40d X %d \n",wbImage_getWidth(inputImage),wbImage_getHeight(inputImage));
   
    //IplImage *img = cvLoadImage("input0.ppm",CV_LOAD_IMAGE_GRAYSCALE);   
    printf("Image Loading time: %40.6lf secs\n",(double)(clock()-begin_imread)/(double)(CLOCKS_PER_SEC));

    maskRows = MASK_SIZE;
    maskColumns = MASK_SIZE; 
    
    float mask[MASK_SIZE][MASK_SIZE];
    float x,y;
    clock_t begin_gauss = clock();
    for(int i=0;i<MASK_SIZE;i++){
	for(int j=0;j<MASK_SIZE;j++){
		x = i - (maskRows/2);
		y = j - (maskColumns/2);
		mask[i][j] = -1.0 * (2 * sigma * sigma - (x * x + y * y)) /(2.0 * PI * sigma * sigma * sigma * sigma) * exp(-(x * x + y * y) / (2.0 * sigma * sigma));				
		hostMaskData[i*MASK_SIZE+j] = mask[i][j];
		}
	}    
     clock_t end_gauss = clock();
     printf("Log Filter execution time: %40.6lf secs\n",(double)(end_gauss-begin_gauss)/(double)(CLOCKS_PER_SEC));

      /*for(int i=0;i<MASK_SIZE;i++){
	
	for(int j=0;j<MASK_SIZE;j++){
		printf("%.1f ",hostMaskData[i*MASK_SIZE+j]);
	}
	cout<<endl;
	}
    */
    //////////////////////////////

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

	
   // Mat A = Mat(imageHeight, imageWidth, CV_32FC3 ,wbImage_getData(inputImage));
    
   // A.convertTo(A, CV_8UC3, 255.0);   
   // imwrite("Wind.jpg",A);
    
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);//(float *)img->imageData;
    hostOutputImageData = wbImage_getData(outputImage);

    clock_t begin_gpu_comp = clock();
    
    clock_t begin_gpu_malloc = clock();
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    
    printf("GPU memory allocation time: %40.6lf secs\n",(double)(clock()-begin_gpu_malloc)/(double)(CLOCKS_PER_SEC));

    clock_t begin_copy_htod = clock();
    hipMemcpyToSymbol(HIP_SYMBOL(M), hostMaskData, sizeof(int) * MASK_SIZE * MASK_SIZE);//
    hipMemcpy(deviceInputImageData, hostInputImageData,imageWidth * imageHeight * imageChannels * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData, hostMaskData,maskRows * maskColumns * sizeof(float),hipMemcpyHostToDevice);

    printf("Copy Time HOST to Device: %40.6lf secs\n",(double)(clock()-begin_copy_htod)/(double)(CLOCKS_PER_SEC));

    
    hipEvent_t start,stop;
    float tot;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    clock_t begin_comp = clock();
    dim3 dimGrid(ceil((float) imageWidth / TILE_WIDTH),ceil((float) imageHeight / TILE_WIDTH));

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    convolution2D<<<dimGrid, dimBlock>>>(deviceInputImageData, /*deviceMaskData,*/deviceOutputImageData, imageChannels, imageWidth, imageHeight);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tot,start,stop);

    printf("Computation time on GPU: %40.6lf secs\n",(double)(clock()-begin_comp)/(double)(CLOCKS_PER_SEC));

    clock_t begin_copy_dtoh = clock();
    hipMemcpy(hostOutputImageData, deviceOutputImageData,imageWidth * imageHeight * imageChannels * sizeof(float),hipMemcpyDeviceToHost);
    printf("Copy time Device to HOST: %40.6lf secs\n",(double)(clock()-begin_copy_dtoh)/(double)(CLOCKS_PER_SEC));
    
    printf("Total time: %40.6lf secs\n",(double)(clock()-begin_gpu_comp)/(double)(CLOCKS_PER_SEC));

    Mat B = Mat(imageHeight, imageWidth, CV_32FC3, wbImage_getData(outputImage));
    B.convertTo(B, CV_8UC3, 255.0);
    imwrite("OUTPUT.jpg",B);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    wbImage_delete(outputImage);
    wbImage_delete(inputImage);
    
    cvWaitKey(0);
    return 0;
}
